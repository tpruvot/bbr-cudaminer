#include "hip/hip_runtime.h"
extern "C" {
#include "miner.h"
}

static hipStream_t *scr_copy_streams;
static ulonglong4 **d_scratchpad;
static uint64_t **d_input;
static uint32_t **d_retnonce;

extern unsigned int CUDABlocks, CUDAThreads;

#define st0 	vst0.x
#define st1 	vst0.y
#define st2 	vst0.z
#define st3 	vst0.w

#define st4 	vst4.x
#define st5 	vst4.y
#define st6 	vst4.z
#define st7 	vst4.w

#define st8 	vst8.x
#define st9 	vst8.y
#define st10	vst8.z
#define st11	vst8.w

#define st12	vst12.x
#define st13	vst12.y
#define st14	vst12.z
#define st15	vst12.w

#define st16	vst16.x
#define st17	vst16.y
#define st18	vst16.z
#define st19	vst16.w

#define st20	vst20.x
#define st21	vst20.y
#define st22	vst20.z
#define st23	vst20.w

__noinline__ __device__ uint64_t bitselect(const uint64_t a, const uint64_t b, const uint64_t c) { return(a ^ (c & (b ^ a))); }
__noinline__ __device__ uint64_t cuda_rotl641(const uint64_t x) { return((x << 1) | (x >> 63)); }

#if __CUDA_ARCH__ >= 320
__device__ __forceinline__ uint64_t cuda_rotl64(const uint64_t value, const int offset)
{
	uint2 result;
	if(offset >= 32) {
		asm("shf.l.wrap.b32 %0, %1, %2, %3;"
			: "=r"(result.x) : "r"(__double2loint(__longlong_as_double(value))), "r"(__double2hiint(__longlong_as_double(value))), "r"(offset));
		asm("shf.l.wrap.b32 %0, %1, %2, %3;"
			: "=r"(result.y) : "r"(__double2hiint(__longlong_as_double(value))), "r"(__double2loint(__longlong_as_double(value))), "r"(offset));
	} else {
		asm("shf.l.wrap.b32 %0, %1, %2, %3;"
			: "=r"(result.x) : "r"(__double2hiint(__longlong_as_double(value))), "r"(__double2loint(__longlong_as_double(value))), "r"(offset));
		asm("shf.l.wrap.b32 %0, %1, %2, %3;"
			: "=r"(result.y) : "r"(__double2loint(__longlong_as_double(value))), "r"(__double2hiint(__longlong_as_double(value))), "r"(offset));
	}
	return __double_as_longlong(__hiloint2double(result.y, result.x));
}
#else
__noinline__ __device__ uint64_t cuda_rotl64(const uint64_t x, const uint8_t y) { return((x << y) | (x >> (64 - y))); }
#endif

#define ROTL64(x, y) (cuda_rotl64((x), (y)))
#define ROTL641(x) (cuda_rotl641(x))

#define RND() \
	bc[0] = st0 ^ st5 ^ st10 * st15 * st20 ^ ROTL641(st2 ^ st7 ^ st12 * st17 * st22); \
	bc[1] = st1 ^ st6 ^ st11 * st16 * st21 ^ ROTL641(st3 ^ st8 ^ st13 * st18 * st23); \
	bc[2] = st2 ^ st7 ^ st12 * st17 * st22 ^ ROTL641(st4 ^ st9 ^ st14 * st19 * st24); \
	bc[3] = st3 ^ st8 ^ st13 * st18 * st23 ^ ROTL641(st0 ^ st5 ^ st10 * st15 * st20); \
	bc[4] = st4 ^ st9 ^ st14 * st19 * st24 ^ ROTL641(st1 ^ st6 ^ st11 * st16 * st21); \
	tmp1 = st1 ^ bc[0]; \
	\
	st0 ^= bc[4]; \
	st1 = ROTL64(st6 ^ bc[0], 44); \
	st6 = ROTL64(st9 ^ bc[3], 20); \
	st9 = ROTL64(st22 ^ bc[1], 61); \
	st22 = ROTL64(st14 ^ bc[3], 39); \
	st14 = ROTL64(st20 ^ bc[4], 18); \
	st20 = ROTL64(st2 ^ bc[1], 62); \
	st2 = ROTL64(st12 ^ bc[1], 43); \
	st12 = ROTL64(st13 ^ bc[2], 25); \
	st13 = ROTL64(st19 ^ bc[3], 8); \
	st19 = ROTL64(st23 ^ bc[2], 56); \
	st23 = ROTL64(st15 ^ bc[4], 41); \
	st15 = ROTL64(st4 ^ bc[3], 27); \
	st4 = ROTL64(st24 ^ bc[3], 14); \
	st24 = ROTL64(st21 ^ bc[0], 2); \
	st21 = ROTL64(st8 ^ bc[2], 55); \
	st8 = ROTL64(st16 ^ bc[0], 45); \
	st16 = ROTL64(st5 ^ bc[4], 36); \
	st5 = ROTL64(st3 ^ bc[2], 28); \
	st3 = ROTL64(st18 ^ bc[2], 21); \
	st18 = ROTL64(st17 ^ bc[1], 15); \
	st17 = ROTL64(st11 ^ bc[0], 10); \
	st11 = ROTL64(st7 ^ bc[1], 6); \
	st7 = ROTL64(st10 ^ bc[4], 3); \
	st10 = ROTL641(tmp1); \
	\
	tmp1 = st0; tmp2 = st1; st0 = bitselect(st0 ^ st2, st0, st1); st1 = bitselect(st1 ^ st3, st1, st2); st2 = bitselect(st2 ^ st4, st2, st3); st3 = bitselect(st3 ^ tmp1, st3, st4); st4 = bitselect(st4 ^ tmp2, st4, tmp1); \
	tmp1 = st5; tmp2 = st6; st5 = bitselect(st5 ^ st7, st5, st6); st6 = bitselect(st6 ^ st8, st6, st7); st7 = bitselect(st7 ^ st9, st7, st8); st8 = bitselect(st8 ^ tmp1, st8, st9); st9 = bitselect(st9 ^ tmp2, st9, tmp1); \
	tmp1 = st10; tmp2 = st11; st10 = bitselect(st10 ^ st12, st10, st11); st11 = bitselect(st11 ^ st13, st11, st12); st12 = bitselect(st12 ^ st14, st12, st13); st13 = bitselect(st13 ^ tmp1, st13, st14); st14 = bitselect(st14 ^ tmp2, st14, tmp1); \
	tmp1 = st15; tmp2 = st16; st15 = bitselect(st15 ^ st17, st15, st16); st16 = bitselect(st16 ^ st18, st16, st17); st17 = bitselect(st17 ^ st19, st17, st18); st18 = bitselect(st18 ^ tmp1, st18, st19); st19 = bitselect(st19 ^ tmp2, st19, tmp1); \
	tmp1 = st20; tmp2 = st21; st20 = bitselect(st20 ^ st22, st20, st21); st21 = bitselect(st21 ^ st23, st21, st22); st22 = bitselect(st22 ^ st24, st22, st23); st23 = bitselect(st23 ^ tmp1, st23, st24); st24 = bitselect(st24 ^ tmp2, st24, tmp1); \
	st0 ^= 1;

#define LASTRND1() \
	bc[0] = st0 ^ st5 ^ st10 * st15 * st20 ^ ROTL64(st2 ^ st7 ^ st12 * st17 * st22, 1); \
	bc[1] = st1 ^ st6 ^ st11 * st16 * st21 ^ ROTL64(st3 ^ st8 ^ st13 * st18 * st23, 1); \
	bc[2] = st2 ^ st7 ^ st12 * st17 * st22 ^ ROTL64(st4 ^ st9 ^ st14 * st19 * st24, 1); \
	bc[3] = st3 ^ st8 ^ st13 * st18 * st23 ^ ROTL64(st0 ^ st5 ^ st10 * st15 * st20, 1); \
	bc[4] = st4 ^ st9 ^ st14 * st19 * st24 ^ ROTL64(st1 ^ st6 ^ st11 * st16 * st21, 1); \
	\
	st0 ^= bc[4]; \
	st1 = ROTL64(st6 ^ bc[0], 44); \
	st2 = ROTL64(st12 ^ bc[1], 43); \
	st4 = ROTL64(st24 ^ bc[3], 14); \
	st3 = ROTL64(st18 ^ bc[2], 21); \
	\
	tmp1 = st0; st0 = bitselect(st0 ^ st2, st0, st1); st1 = bitselect(st1 ^ st3, st1, st2); st2 = bitselect(st2 ^ st4, st2, st3); st3 = bitselect(st3 ^ tmp1, st3, st4); \
	st0 ^= 1;

#define LASTRND2() \
	bc[2] = st2 ^ st7 ^ st12 * st17 * st22 ^ ROTL64(st4 ^ st9 ^ st14 * st19 * st24, 1); \
	bc[3] = st3 ^ st8 ^ st13 * st18 * st23 ^ ROTL64(st0 ^ st5 ^ st10 * st15 * st20, 1); \
	bc[4] = st4 ^ st9 ^ st14 * st19 * st24 ^ ROTL64(st1 ^ st6 ^ st11 * st16 * st21, 1); \
	\
	st0 ^= bc[4]; \
	st4 = ROTL64(st24 ^ bc[3], 14); \
	st3 = ROTL64(st18 ^ bc[2], 21); \
	st3 = bitselect(st3 ^ st0, st3, st4);

__device__ ulonglong4 operator^(const ulonglong4 &a, const ulonglong4 &b)
{
	return(make_ulonglong4(a.x ^ b.x, a.y ^ b.y, a.z ^ b.z, a.w ^ b.w));
}

#define MIX(vst) vst = vst ^ scratchpad[vst.x % scr_size] ^ scratchpad[vst.y % scr_size] ^ scratchpad[vst.z % scr_size] ^ scratchpad[vst.w % scr_size];

#define MIX_ALL MIX(vst0); MIX(vst4); MIX(vst8); MIX(vst12); MIX(vst16); MIX(vst20);

__global__
void wk(uint32_t * __restrict__ retnonce, const uint64_t * __restrict__ input, const ulonglong4 * __restrict__ scratchpad, const uint32_t scr_size, uint64_t nonce, const uint32_t target)
{
	ulonglong4 vst0, vst4, vst8, vst12, vst16, vst20;
	uint64_t __restrict__ bc[5], st24, tmp1, tmp2;

	nonce += (blockDim.x * blockIdx.x) + threadIdx.x;

	vst0 	= make_ulonglong4((nonce << 8) + (input[0] & 0xFF), input[1] & 0xFFFFFFFFFFFFFF00ULL, input[2], input[3]);
	vst4 	= make_ulonglong4(input[4], input[5], input[6], input[7]);
	vst8 	= make_ulonglong4(input[8], input[9], (input[10] & 0xFF) | 0x100, 0);
	vst12 	= make_ulonglong4(0, 0, 0, 0);
	vst16 	= make_ulonglong4(0x8000000000000000ULL, 0, 0, 0);
	vst20	= make_ulonglong4(0, 0, 0, 0);
	st24 	= 0;

	RND();
	MIX_ALL;

	for(int i = 0; i < 22; ++i)
	{
		RND();
		MIX_ALL;
	}

	LASTRND1();

	vst4 	= make_ulonglong4(1, 0, 0, 0);
	vst8 	= make_ulonglong4(0, 0, 0, 0);
	vst12 	= make_ulonglong4(0, 0, 0, 0);
	vst16	= make_ulonglong4(0x8000000000000000ULL, 0, 0, 0);
	vst20	= make_ulonglong4(0, 0, 0, 0);
	st24	= 0;

	RND();
	MIX_ALL;

	for(int i = 0; i < 22; ++i)
	{
		RND();
		MIX_ALL;
	}

	LASTRND2();

	if((st3 >> 32) <= target) *retnonce = (uint32_t)nonce;
}

extern "C" void UpdateScratchpad(uint32_t threads)
{
	for(int i = 0; i < threads; ++i)
		hipMemcpyAsync(d_scratchpad[i], pscratchpad_buff, scratchpad_size << 3, hipMemcpyHostToDevice, scr_copy_streams[i]);
}

extern "C" void InitCUDA(uint32_t threads, char **devstrs)
{
	struct hipDeviceProp_t prop;
	int numdevs;

	if(hipGetDeviceCount(&numdevs) != hipSuccess)
	{
		applog(LOG_ERR, "Something's fucked - can't get number of CUDA devices.");
		exit(0);
	}

	if(threads > numdevs)
	{
		applog(LOG_ERR, "You specified more threads than there are CUDA devices, you idiot.");
		exit(0);
	}

	scr_copy_streams = (hipStream_t *)malloc(sizeof(hipStream_t) * threads);

	d_scratchpad = (ulonglong4 **)malloc(sizeof(ulonglong4 *) * threads);
	d_input = (uint64_t **)malloc(sizeof(uint64_t *) * threads);
	d_retnonce = (uint32_t **)malloc(sizeof(uint32_t *) * threads);

	for(int i = 0; i < threads; ++i)
	{
		hipGetDeviceProperties(&prop, i);
		devstrs[i] = strdup(prop.name);
	}

}

extern "C" void CUDASetDevice(uint32_t thread_id)
{
	int i = (int) thread_id;
	hipSetDevice(i);
	hipDeviceReset();
	hipSetDeviceFlags(hipDeviceScheduleBlockingSync);

	#ifdef USE_MAPPED_MEMORY
	hipSetDeviceFlags(hipDeviceMapHost);
	hipDeviceSetCacheConfig(hipFuncCachePreferL1);
	#endif

	hipMalloc(&d_scratchpad[i], WILD_KECCAK_SCRATCHPAD_BUFFSIZE);

#ifdef USE_MAPPED_MEMORY
	hipHostAlloc(&d_retnonce[i], sizeof(uint32_t), hipHostMallocMapped);
#else
	hipMalloc(&d_retnonce[i], sizeof(uint32_t));
#endif
	hipMalloc(&d_input[i], 88);
	hipStreamCreate(&scr_copy_streams[i]);
}

extern "C" int scanhash_wildkeccak(int thr_id, uint32_t *pdata, const uint32_t *ptarget, uint32_t max_nonce, unsigned long *hashes_done)
{
	uint32_t *nonceptr = ((uint32_t *)(((uint8_t *)pdata) + 1));
	uint32_t n = *nonceptr;
	uint32_t first = n, blocks = CUDABlocks, threads = CUDAThreads;

	hipMemcpy(d_input[thr_id], pdata, 88, hipMemcpyHostToDevice);

#ifdef USE_MAPPED_MEMORY
	*(d_retnonce[thr_id]) = 0xFFFFFFFFUL;
	uint32_t *dnonce;
	hipHostGetDevicePointer(&dnonce, d_retnonce[thr_id], 0);
#else
	hipMemset(d_retnonce[thr_id], 0xFF, sizeof(uint32_t));
	uint32_t h_retnonce;
#endif

	hipStreamSynchronize(scr_copy_streams[thr_id]);

	do
	{
		dim3 block(blocks);
		dim3 thread(threads);

#ifdef USE_MAPPED_MEMORY
		wk<<<block, thread, 0, scr_copy_streams[thr_id]>>>(dnonce, d_input[thr_id], d_scratchpad[thr_id], (uint32_t)(scratchpad_size >> 2), n, ptarget[7]);
		//hipDeviceSynchronize();
		if(*(d_retnonce[thr_id]) < 0xFFFFFFFFU)
		{
			*nonceptr = *(d_retnonce[thr_id]);
			*hashes_done = *(d_retnonce[thr_id]) - first + 1;
			return(1);
		}
#else
		wk<<<block, thread, 0, scr_copy_streams[thr_id]>>>(d_retnonce[thr_id], d_input[thr_id], d_scratchpad[thr_id], (uint32_t)(scratchpad_size >> 2), n, ptarget[7]);
		//hipDeviceSynchronize();
		hipMemcpy(&h_retnonce, d_retnonce[thr_id], sizeof(uint32_t), hipMemcpyDeviceToHost);
		if(h_retnonce < 0xFFFFFFFFU)
		{
			*nonceptr = h_retnonce;
			*hashes_done = h_retnonce - first + 1;
			return(1);
		}
#endif

		n += blocks * threads;
	} while(n < max_nonce && !work_restart[thr_id].restart);

	*hashes_done = n - first + 1;
	return(0);
}
